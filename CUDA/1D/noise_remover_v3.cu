#include "hip/hip_runtime.h"
/*	
 * noise_remover.cpp
 *
 * This program removes noise from an h_image based on Speckle Reducing Anisotropic Diffusion
 * Y. Yu, S. Acton, Speckle reducing anisotropic diffusion, 
 * IEEE Transactions on h_image Processing 11(11)(2002) 1260-1270 <http://people.virginia.edu/~sc5nf/01097762.pdf>
 * Original implementation is Modified by Burak BASTEM
 */
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <string.h>
#include <sys/time.h>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
const int threadsPerBlock = 512;
#define MATCH(s) (!strcmp(argv[ac], (s)))

// returns the current time
static const double kMicro = 1.0e-6;
double get_time() {
	struct timeval TV;
	struct timezone TZ;
	const int RC = gettimeofday(&TV, &TZ);
	if(RC == -1) {
		printf("ERROR: Bad call to gettimeofday\n");
		return(-1);
	}
	return( ((double)TV.tv_sec) + kMicro * ((double)TV.tv_usec) );
}

__global__ void
reductionAndStatistics(unsigned char *d_image, double *sum, double *sum2, int imageSize) {
	__shared__ double d_sum[threadsPerBlock];
	__shared__ double d_sum2[threadsPerBlock];
	long blockSize = blockDim.x * blockDim.y;
	long offset = threadIdx.x + blockIdx.x * threadsPerBlock;
	int cacheIndex = threadIdx.x;
	if (offset < imageSize) {
		double tmp = d_image[offset];
		d_sum[cacheIndex] = tmp; // --- 1 doubleing point arithmetic operations
		d_sum2[cacheIndex] = tmp*tmp; // --- 2 doubleing point arithmetic operations
		__syncthreads();
		if (blockSize >= 512) {
			if (cacheIndex < 256) {
				d_sum[cacheIndex] += d_sum[cacheIndex + 256];
				d_sum2[cacheIndex] += d_sum2[cacheIndex + 256];				
			}
			__syncthreads();
		}

		if (blockSize >= 256) {
			if (cacheIndex < 128) {
				d_sum[cacheIndex] += d_sum[cacheIndex + 128];
				d_sum2[cacheIndex] += d_sum2[cacheIndex + 128];				
			}
			__syncthreads();
		}

		if (blockSize >= 128) {
			if (cacheIndex < 64) {
				d_sum[cacheIndex] += d_sum[cacheIndex + 64];
				d_sum2[cacheIndex] += d_sum2[cacheIndex + 64];				
			}
			__syncthreads();
		}

		if (cacheIndex < 32) {	
			if (blockSize >= 64)  {
				d_sum[cacheIndex] += d_sum[cacheIndex + 32];
				d_sum2[cacheIndex] += d_sum2[cacheIndex + 32]; 
			}
			if (blockSize >= 32)  {
				d_sum[cacheIndex] += d_sum[cacheIndex + 16];
				d_sum2[cacheIndex] += d_sum2[cacheIndex + 16]; 
			}
			if (blockSize >= 16) {
				d_sum[cacheIndex] += d_sum[cacheIndex + 8];
				d_sum2[cacheIndex] += d_sum2[cacheIndex + 8]; 
			}
			if (blockSize >= 8) {
				d_sum[cacheIndex] += d_sum[cacheIndex + 4];
				d_sum2[cacheIndex] += d_sum2[cacheIndex + 4]; 
			}
			if (blockSize >= 4) {
				d_sum[cacheIndex] += d_sum[cacheIndex + 2];
				d_sum2[cacheIndex] += d_sum2[cacheIndex + 2]; 
			}
			if (blockSize >= 2) {
				d_sum[cacheIndex] += d_sum[cacheIndex + 1];
				d_sum2[cacheIndex] += d_sum2[cacheIndex + 1]; 
			}
		}
	}

	if (cacheIndex == 0) {
		sum[blockIdx.x] = d_sum[0];
		sum2[blockIdx.x] = d_sum2[0];
	}
}

__global__ void
Compute2(unsigned char *d_image, double *d_north_deriv, double *d_south_deriv, double *d_east_deriv, double *d_west_deriv, double *d_diff_coef, double d_lambda, int width, int height) {
	long threadsPerRow = width;
	long totalThreads = height*width;
	long k = threadIdx.x + blockIdx.x * threadsPerBlock;	// get position of current element
	double diff_coef_north, diff_coef_south, diff_coef_west, diff_coef_east, divergence;	// directional diffusion coefficients
	double d_diff_coef_register;
    
    if (k >= threadsPerRow && k < totalThreads-threadsPerRow) {
    	if (((k % threadsPerRow) != 0) && ((k+1) % threadsPerRow) != 0) {
    		d_diff_coef_register = d_diff_coef[k];
			diff_coef_north = d_diff_coef_register;						// north diffusion coefficient
		    diff_coef_south = d_diff_coef[k+threadsPerRow];			// south diffusion coefficient
	    	diff_coef_west = d_diff_coef_register;						// west diffusion coefficient
	    	diff_coef_east = d_diff_coef[k + 1];					// east diffusion coefficient

	    	divergence = diff_coef_north * d_north_deriv[k] + diff_coef_south * 
	    				d_south_deriv[k] + diff_coef_west * d_west_deriv[k] + diff_coef_east * d_east_deriv[k]; // --- 7 doubleing point arithmetic operations
	    	d_image[k] = d_image[k] + 0.25f * d_lambda * divergence; // --- 3 doubleing point arithmetic operations
	    }
    }
}

__global__ void
Compute1(unsigned char *d_image, double std_dev, double *d_north_deriv, double *d_south_deriv, double *d_east_deriv, double *d_west_deriv, double *d_diff_coef, int width, int height, int blocksPerGrid) {
	long k = threadIdx.x + blockIdx.x * threadsPerBlock;
	long totalThreads = height*width;	//17915904
	long threadsPerRow = width;
	double d_image_temp = d_image[k];
	double gradient_square, laplacian, num, den, std_dev2;	// calculation variables
	double north_deriv_value, south_deriv_value, east_deriv_value, west_deriv_value;
	__shared__ unsigned char img_data[3][threadsPerBlock];
	__shared__ unsigned char img_data_before, img_data_after;
	unsigned char img_data_register;	
	
	if (k >= threadsPerRow && k < totalThreads-threadsPerRow) {
		if (threadIdx.x == 0) {
			img_data_before = d_image[(blockIdx.x-1)*threadsPerBlock + (threadsPerBlock - 1)];
		}

		if (threadIdx.x == threadsPerBlock-1) {
			img_data_after = d_image[(blockIdx.x+1)*threadsPerBlock];
		}

		img_data[0][threadIdx.x] = d_image[k - threadsPerRow];
		img_data[1][threadIdx.x] = d_image[k];
		img_data[2][threadIdx.x] = d_image[k + threadsPerRow];
		__syncthreads();

		if (((k % threadsPerRow) != 0) && ((k+1) % threadsPerRow) != 0) {
			img_data_register = img_data[1][threadIdx.x];

			// north derivative --- 1 doubleing point arithmetic operations
			d_north_deriv[k] = img_data[0][threadIdx.x] - img_data_register;	
			north_deriv_value = d_north_deriv[k];

			// south derivative --- 1 doubleing point arithmetic operations
			d_south_deriv[k] = img_data[2][threadIdx.x] - img_data_register;	
			south_deriv_value = d_south_deriv[k];

			// west derivative --- 1 doubleing point arithmetic operations
			if (threadIdx.x != 0) {
				d_west_deriv[k] = img_data[1][threadIdx.x - 1] - img_data_register;	
			} else {
				d_west_deriv[k] = img_data_before - img_data_register;
			}
			west_deriv_value = d_west_deriv[k];

			// east derivative --- 1 doubleing point arithmetic operations
			if (threadIdx.x != threadsPerBlock - 1) {
				d_east_deriv[k] = img_data[1][threadIdx.x + 1] - img_data_register;	
			} else {
				d_east_deriv[k] = img_data_after - img_data_register;
			}
			east_deriv_value = d_east_deriv[k];


			gradient_square = (north_deriv_value * north_deriv_value + south_deriv_value * south_deriv_value + 
								west_deriv_value * west_deriv_value + east_deriv_value * east_deriv_value) 
								/ (d_image_temp * d_image_temp); // 9 doubleing point arithmetic operations

			laplacian = (north_deriv_value + south_deriv_value + west_deriv_value + east_deriv_value) / d_image_temp; // 4 doubleing point arithmetic operations
			
			num = (0.5f * gradient_square) - ((1.0f / 16.0f) * (laplacian * laplacian)); // 5 doubleing point arithmetic operations
			den = 1 + (.25f * laplacian); // 2 doubleing point arithmetic operations
			std_dev2 = num / (den * den); // 2 doubleing point arithmetic operations
			den = (std_dev2 - std_dev) / (std_dev * (1 + std_dev)); // 4 doubleing point arithmetic operations
			d_diff_coef[k] = 1.0f / (1.0f + den); // 2 doubleing point arithmetic operations
			if (d_diff_coef[k] < 0) {
				d_diff_coef[k] = 0;
			} else if (d_diff_coef[k] > 1)	{
				d_diff_coef[k] = 1;
			}
			// __syncthreads();
		}
	}
}


void getGPUInfo() {
	hipDeviceProp_t prop;
	int count;
	hipGetDeviceCount(&count);
	printf("Installed GPUs: %d\n", count);
	for (int i=0; i< count; i++) {
		hipGetDeviceProperties(&prop, i);
		printf( " --- General Information for device %d ---\n", i );
		printf( "Name: %s\n", prop.name );
		printf( "Compute capability: %d.%d\n", prop.major, prop.minor );
		printf( "Clock rate: %d\n", prop.clockRate );
		printf( "Device copy overlap: " );
		if (prop.deviceOverlap)
		printf( "Enabled\n" );
		else
		printf( "Disabled\n" );
		printf( "Kernel execition timeout : " );
		if (prop.kernelExecTimeoutEnabled)
		printf( "Enabled\n" );
		else
		printf( "Disabled\n" );
		printf( " --- Memory Information for device %d ---\n", i );
		printf( "Total global mem: %ld\n", prop.totalGlobalMem );
		printf( "Total constant Mem: %ld\n", prop.totalConstMem );
		printf( "Max mem pitch: %ld\n", prop.memPitch );
		printf( "Texture Alignment: %ld\n", prop.textureAlignment );
		printf( " --- MP Information for device %d ---\n", i );
		printf( "Multiprocessor count: %d\n",
		prop.multiProcessorCount );
		printf( "Shared mem per mp: %ld\n", prop.sharedMemPerBlock );
		printf( "Registers per mp: %d\n", prop.regsPerBlock );
		printf( "Threads in warp: %d\n", prop.warpSize );
		printf( "Max threads per block: %d\n",
		prop.maxThreadsPerBlock );
		printf( "Max thread dimensions: (%d, %d, %d)\n",
		prop.maxThreadsDim[0], prop.maxThreadsDim[1],
		prop.maxThreadsDim[2] );
		printf( "Max grid dimensions: (%d, %d, %d)\n",
		prop.maxGridSize[0], prop.maxGridSize[1],
		prop.maxGridSize[2] );
		printf( "\n" );
	}
}

int main(int argc, char *argv[]) {
	// Part I: allocate and initialize variables
	double time_0, time_1, time_2, time_3, time_4, time_5, time_6, time_7, time_8;	// time variables
	time_0 = get_time();
	const char *filename = "input.pgm";
	const char *outputname = "output.png";	
	
	int n_iter = 50;
	double lambda = 0.5;
	double mean, variance, std_dev;	//local region statistics
	double *north_deriv, *south_deriv, *west_deriv, *east_deriv;	// directional derivatives
	double tmp, sum, sum2;	// calculation variables
	double gradient_square, laplacian, num, den, std_dev2, divergence;	// calculation variables
	double *diff_coef;	// diffusion coefficient
	double diff_coef_north, diff_coef_south, diff_coef_west, diff_coef_east;	// directional diffusion coefficients
	long k;	// current pixel index
	int width = 0;
	int height = 0;
	int pixelWidth = 0;
	int n_pixels = 0;
	int blocksPerGrid = 0;
	time_1 = get_time();
	
	// Part II: parse command line arguments
	if(argc<2) {
	  printf("Usage: %s [-i < filename>] [-iter <n_iter>] [-l <lambda>] [-o <outputfilename>]\n",argv[0]);
	  return(-1);
	}
	for(int ac=1;ac<argc;ac++) {
		if(MATCH("-i")) {
			filename = argv[++ac];
		} else if(MATCH("-iter")) {
			n_iter = atoi(argv[++ac]);
		} else if(MATCH("-l")) {
			lambda = atof(argv[++ac]);
		} else if(MATCH("-o")) {
			outputname = argv[++ac];
		} else {
		printf("Usage: %s [-i < filename>] [-iter <n_iter>] [-l <lambda>] [-o <outputfilename>]\n",argv[0]);
		return(-1);
		}
	}
	time_2 = get_time();

	// CUDA variables
	// Error code: to check return values for CUDA calls
    hipError_t err = hipSuccess;
    // getGPUInfo();

	// Part III: read h_image	
	printf("Reading h_image...\n");
	unsigned char *h_image = stbi_load(filename, &width, &height, &pixelWidth, 0);
	size_t size = height*width;
    blocksPerGrid = (size + (threadsPerBlock - 1))/threadsPerBlock;

	unsigned char *d_image = NULL;
	err = hipMalloc((void **)&d_image, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (Error code: %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	if (!h_image) {
		fprintf(stderr, "Couldn't load h_image.\n");
		return (-1);
	}
	// printf("h_image Read. Width : %d, Height : %d, nComp: %d\n",width,height,pixelWidth);
	n_pixels = height * width;
	time_3 = get_time();

	// Part IV: allocate variables
	north_deriv = (double*) malloc(sizeof(double) * n_pixels);	// north derivative
	south_deriv = (double*) malloc(sizeof(double) * n_pixels);	// south derivative
	west_deriv = (double*) malloc(sizeof(double) * n_pixels);	// west derivative
	east_deriv = (double*) malloc(sizeof(double) * n_pixels);	// east derivative
	diff_coef  = (double*) malloc(sizeof(double) * n_pixels);	// diffusion coefficient

	double *d_partial_sum, *d_partial_sum2;
	hipMalloc((void**)&d_partial_sum, blocksPerGrid*sizeof(double));
	hipMalloc((void**)&d_partial_sum2, blocksPerGrid*sizeof(double));

	double *partial_sum = (double*)malloc(blocksPerGrid*sizeof(double));   
	double *partial_sum2 = (double*)malloc(blocksPerGrid*sizeof(double));

	time_4 = get_time();
	// Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    // printf("Copy h_image from the host memory to the CUDA device\n");
    err = hipMemcpy(d_image, h_image, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy h_image from host to device (Error code:: %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Compute 1: Allocating variables for device 
	double *d_north_deriv, *d_south_deriv, *d_west_deriv, *d_east_deriv, *d_diff_coef;
	hipMalloc((void **)&d_north_deriv, sizeof(double) * n_pixels);	// north derivative
	hipMalloc((void **)&d_south_deriv, sizeof(double) * n_pixels);	// south derivative
	hipMalloc((void **)&d_west_deriv, sizeof(double) * n_pixels);	// west derivative
	hipMalloc((void **)&d_east_deriv, sizeof(double) * n_pixels);	// east derivative
	hipMalloc((void **)&d_diff_coef, sizeof(double) * n_pixels);	// diffusion coefficient

	// Part V: compute --- n_iter * (3 * height * width + 42 * (height-1) * (width-1) + 6) doubleing point arithmetic operations in totaL
	for (int iter = 0; iter < n_iter; iter++) {
		sum = 0;
		sum2 = 0;

		// Kernel Launch	
		reductionAndStatistics<<<blocksPerGrid, threadsPerBlock>>>(d_image, d_partial_sum, d_partial_sum2, height*width);
		err = hipGetLastError();
	    if (err != hipSuccess) {
	        fprintf(stderr, "Failed to launch reductionAndStatistics kernel (Error code: %s)!\n", hipGetErrorString(err));
	        exit(EXIT_FAILURE);
	    }

	    hipMemcpy(partial_sum, d_partial_sum, blocksPerGrid*sizeof(double), hipMemcpyDeviceToHost);
	    hipMemcpy(partial_sum2, d_partial_sum2, blocksPerGrid*sizeof(double), hipMemcpyDeviceToHost);
	    
	    // hipDeviceSynchronize();

	    for (int i = 0 ; i < blocksPerGrid ; i++) {
	    	sum += partial_sum[i];
	    	sum2 += partial_sum2[i];
	    }

	    // printf("Sum: %f Sum2: %f\n", sum, sum2);

		mean = sum / n_pixels; // --- 1 doubleing point arithmetic operations
		variance = (sum2 / n_pixels) - mean * mean; // --- 3 doubleing point arithmetic operations
		std_dev = variance / (mean * mean); // --- 2 doubleing point arithmetic operations

		// Compute 1: Kernel Launch 
		Compute1<<<blocksPerGrid, threadsPerBlock>>>(d_image, std_dev, d_north_deriv, d_south_deriv, d_west_deriv, d_east_deriv, d_diff_coef, width, height, blocksPerGrid);
		err = hipGetLastError();
	    if (err != hipSuccess) {
	        fprintf(stderr, "Failed to launch Compute1 kernel (Error code: %s)!\n", hipGetErrorString(err));
	        exit(EXIT_FAILURE);
	    }

	    hipDeviceSynchronize();

	    // Compute 2: Kernel launch
	    Compute2<<<blocksPerGrid, threadsPerBlock>>>(d_image, d_north_deriv, d_south_deriv, d_west_deriv, d_east_deriv, d_diff_coef, lambda, width, height);
		err = hipGetLastError();
	    if (err != hipSuccess) {
	        fprintf(stderr, "Failed to launch Compute2 kernel (Error code: %s)!\n", hipGetErrorString(err));
	        exit(EXIT_FAILURE);
	    }

	    hipDeviceSynchronize();

	}
	err = hipMemcpy(h_image, d_image, size, hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
	    fprintf(stderr, "Failed to copy h_image from host to device (Error code:: %s)!\n", hipGetErrorString(err));
	    exit(EXIT_FAILURE);
	}
	time_5 = get_time();

	// Part VI: write h_image to file
	stbi_write_png(outputname, width, height, pixelWidth, h_image, 0);
	time_6 = get_time();

	// Part VII: get average of sum of pixels for testing and calculate GFLOPS
	// FOR VALIDATION - DO NOT PARALLELIZE
	double test = 0;
	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			test += h_image[i * width + j];
		}
	}
	test /= n_pixels;	

	double gflops = (double) (n_iter * 1E-9 * (3 * height * width + 42 * (height-1) * (width-1) + 6)) / (time_5 - time_4);
	time_7 = get_time();

	// Part VII: deallocate variables
	stbi_image_free(h_image);
	free(north_deriv);
	free(south_deriv);
	free(west_deriv);
	free(east_deriv);
	free(diff_coef);
	hipFree(d_image);
	hipFree(d_north_deriv);
	hipFree(d_south_deriv);
	hipFree(d_west_deriv);
	hipFree(d_east_deriv);
	hipFree(d_diff_coef);
	hipFree(d_partial_sum);
	hipFree(d_partial_sum2);
	time_8 = get_time();

	// print
	printf("Time spent in different stages of the application:\n");
	printf("%9.6f s => Part I: allocate and initialize variables\n", (time_1 - time_0));
	printf("%9.6f s => Part II: parse command line arguments\n", (time_2 - time_1));
	printf("%9.6f s => Part III: read h_image\n", (time_3 - time_2));
	printf("%9.6f s => Part IV: allocate variables\n", (time_4 - time_3));
	printf("%9.6f s => Part V: compute\n", (time_5 - time_4));
	printf("%9.6f s => Part VI: write h_image to file\n", (time_6 - time_5));
	printf("%9.6f s => Part VII: get average of sum of pixels for testing and calculate GFLOPS\n", (time_7 - time_6));
	printf("%9.6f s => Part VIII: deallocate variables\n", (time_7 - time_6));
	printf("Total time: %9.6f s\n", (time_8 - time_0));
	printf("Average of sum of pixels: %9.6f\n", test);
	printf("GFLOPS: %f\n", gflops);
	return 0;
}

